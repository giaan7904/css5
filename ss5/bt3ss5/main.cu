
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int soPhut, cuocPhi;
    const int phiThueBao = 25000;
    printf("Nhap so phut su dung: ");
    scanf("%d", &soPhut);

    if(soPhut <= 50) {
        cuocPhi = phiThueBao + soPhut * 600;
    } else if(soPhut <= 200) {
        cuocPhi = phiThueBao + 50 * 600 + (soPhut - 50) * 400;
    } else if(soPhut <= 400) {
        cuocPhi = phiThueBao + 50 * 600 + 150 * 400 + (soPhut - 200) * 200;
    } else {
        cuocPhi = phiThueBao + 50 * 600 + 150 * 400 + 200 * 200 + (soPhut - 400) * 100;
    }

    printf("Cuoc phi dien thoai ban la: %d VND\n", cuocPhi);

    return 0;
}
